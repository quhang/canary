#include "cuda_helper.h"

#include <hip/hip_runtime.h>
#include "unsupported/Eigen/CXX11/Tensor"

namespace canary {

template<typename T, size_t Dimension>
void GpuTensorStore<T, Dimension>::LoadFromHostVector(
    const std::vector<T>& input) {
  if (input.size() != get_num_elements()) {
    fprintf(stderr,
            "Deserialization for the GpuTensorStore failed internally!\n");
  } else {
    hipMemcpy(data_, input.data(), input.size() * sizeof(T),
               hipMemcpyHostToDevice);
    hipDeviceSynchronize();
  }
}

template<typename T, size_t Dimension>
void GpuTensorStore<T, Dimension>::SaveToHostVector(
    std::vector<T>* input) const {
  if (data_) {
    input->resize(get_num_elements(), 0);
    hipMemcpy(input->data(), data_, input->size() * sizeof(T),
               hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
}

template<typename T, size_t Dimension>
void GpuTensorStore<T, Dimension>::Reset() {
  if (data_) {
    hipFree(data_);
  }
  data_ = nullptr;
  for (auto& elem : ranks_) { elem = 0; }
}

template<typename T, size_t Dimension>
bool GpuTensorStore<T, Dimension>::Allocate(size_t num_elements) {
  return hipMalloc(&data_, num_elements * sizeof(T)) == 0;
}

template class GpuTensorStore<float, 1>;
template class GpuTensorStore<float, 2>;
template class GpuTensorStore<float, 3>;
template class GpuTensorStore<double, 1>;
template class GpuTensorStore<double, 2>;
template class GpuTensorStore<double, 3>;

}  // namespace canary
