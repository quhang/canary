#include "cuda_helper.h"

#include <hip/hip_runtime.h>
#include "unsupported/Eigen/CXX11/Tensor"

namespace canary {

template<typename T, size_t Dimension>
void GpuTensorStore<T, Dimension>::LoadFromHostVector(
    const std::vector<T>& input) {
  if (input.size() != get_num_elements()) {
    fprintf(stderr,
            "Deserialization for the GpuTensorStore failed internally!\n");
  } else {
    hipMemcpy(data_, input.data(), input.size(), hipMemcpyHostToDevice);
  }
}

template<typename T, size_t Dimension>
void GpuTensorStore<T, Dimension>::SaveToHostVector(
    std::vector<T>* input) const {
  if (data_) {
    input->resize(get_num_elements(), 0);
    hipMemcpy(input->data(), data_, input->size() * sizeof(T),
               hipMemcpyDeviceToHost);
  }
}

template<typename T, size_t Dimension>
void GpuTensorStore<T, Dimension>::Reset() {
  if (data_) {
    hipFree(data_);
  }
  data_ = nullptr;
  for (int& elem : ranks_) { elem = 0; }
}

template<typename T, size_t Dimension>
bool GpuTensorStore<T, Dimension>::Allocate(size_t num_elements) {
  return hipMalloc(&data_, num_elements * sizeof(T)) == 0;
}


}  // namespace canary
