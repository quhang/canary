#include "hip/hip_runtime.h"
#define EIGEN_USE_GPU

#include "cuda_helper.h"

#include <hip/hip_runtime.h>
#include "unsupported/Eigen/CXX11/Tensor"
#include "unsupported/Eigen/CXX11/Tensor"

#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>

#include "hipblas.h"

namespace canary {

// The global variable that saves the cuBLAS handler.
hipblasHandle_t cublas_handle;
bool cublas_initialized = false;
// The global variable that saves the GPU device handler.
Eigen::CudaStreamDevice hip_stream(0);
Eigen::GpuDevice gpu_device(&hip_stream);

template<typename T, size_t Dimension>
void GpuTensorStore<T, Dimension>::LoadFromHostVector(
    const std::vector<T>& input) {
  if (input.size() != get_num_elements()) {
    fprintf(stderr,
            "Deserialization for the GpuTensorStore failed internally!\n");
  } else {
    hipMemcpy(data_, input.data(), input.size() * sizeof(T),
               hipMemcpyHostToDevice);
    hipDeviceSynchronize();
  }
}

template<typename T, size_t Dimension>
void GpuTensorStore<T, Dimension>::SaveToHostVector(
    std::vector<T>* input) const {
  if (data_) {
    input->resize(get_num_elements(), 0);
    hipMemcpy(input->data(), data_, input->size() * sizeof(T),
               hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
}

template<typename T, size_t Dimension>
void GpuTensorStore<T, Dimension>::Reset() {
  if (data_) {
    hipFree(data_);
  }
  data_ = nullptr;
  true_size_ = 0;
}

template<typename T, size_t Dimension>
bool GpuTensorStore<T, Dimension>::Allocate(size_t num_elements) {
  if (true_size_ >= num_elements * sizeof(T)) {
    return true;
  } else {
    Reset();
    if (hipMalloc(&data_, num_elements * sizeof(T)) == 0) {
      true_size_ = num_elements * sizeof(T);
      return true;
    } else {
      return false;
    }
  }
}

template class GpuTensorStore<float, 1>;
template class GpuTensorStore<float, 2>;
template class GpuTensorStore<float, 3>;
template class GpuTensorStore<double, 1>;
template class GpuTensorStore<double, 2>;
template class GpuTensorStore<double, 3>;

namespace app {

struct PseudoRandomGenerator {
 double low_, high_;
 __host__ __device__ PseudoRandomGenerator(double low, double high) : low_(low), high_(high) {};
 __host__ __device__ float operator()(const unsigned int n) const {
   thrust::default_random_engine rng;
   thrust::uniform_real_distribution<double> dist(low_, high_);
   rng.discard(n);
   return dist(rng);
  }
};

/*
 * Generate random features and their labels.
 */
void GenerateRandomData(const std::vector<double> reference,
                        GpuTensorStore<double, 2>* x_data,
                        GpuTensorStore<double, 1>* y_data) {
  // Initialize cuBLAS here.
  if (!cublas_initialized) {
    cublas_initialized = true;
    hipblasCreate(&cublas_handle);
  }
  const size_t dim = x_data->get_ranks()[0];
  const size_t samples = x_data->get_ranks()[1];
  if (reference.size() != dim || y_data->get_ranks()[0] != samples) {
    fprintf(stderr, "Mismatched dimensions in GenerateRandomData!\n");
    return;
  }
  Eigen::TensorMap<Eigen::Tensor<double, 2>> x_tensor((double*)x_data->get_data(), dim, samples);
  // The Eigen library cannot generate random numbers correctly.
  // x_tensor.device(gpu_device) = x_tensor.random() - x_tensor.constant(0.5);
  // Instead, use thrust to generate random numbers.
  thrust::counting_iterator<unsigned int> index_sequence_begin(0);
  thrust::transform(index_sequence_begin, index_sequence_begin + dim * samples,
		    thrust::device_ptr<double>((double*)x_data->get_data()),
                    PseudoRandomGenerator(-0.5, 0.5));
  Eigen::TensorMap<Eigen::Tensor<double, 1>> y_tensor((double*)y_data->get_data(), samples);
  GpuTensorStore<double, 1> w_data;
  w_data.ToDevice(reference);
  Eigen::TensorMap<Eigen::Tensor<double, 2>> w_tensor((double*)w_data.get_data(), dim, 1);
  // Call into the Eigen library to compute the labels.
  Eigen::array<int, 2> bcast1({1, int(samples)});
  Eigen::array<int, 1> dims1({0});
  y_tensor.device(gpu_device) =
    ((w_tensor.broadcast(bcast1) * x_tensor).sum(dims1) > y_tensor.constant(0)).select(
        y_tensor.constant(1), y_tensor.constant(-1));
  // Wait for GPU computations to complete.
  hipDeviceSynchronize();
}

// Unused kernels.
// // General version with no assumptiong about dim.
// #if __CUDA_ARCH__ < 600
// __device__ double atomicAdd(double* address, double val)
// {
//     unsigned long long int* address_as_ull =
//                               (unsigned long long int*)address;
//     unsigned long long int old = *address_as_ull, assumed;
// 
//     do {
//         assumed = old;
//         old = atomicCAS(address_as_ull, assumed,
//                         __double_as_longlong(val +
//                                __longlong_as_double(assumed)));
// 
//     // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
//     } while (assumed != old);
// 
//     return __longlong_as_double(old);
// }
// #endif
// 
// __global__ void ComputeDotProduct(double* w_data, double* x_data, double* factor_data, int dim) {
//   if (threadIdx.x == 0) {
//     factor_data[blockIdx.x] = 0;
//   }
//   __syncthreads();
//   int result = w_data[threadIdx.x] * x_data[blockIdx.x * dim + threadIdx.x];
//   // Synchronous instructions within a warp to reduce the sum.
//   // Assume WARP_SIZE = 32.
//   result += __shfl_down(result, 16);
//   result += __shfl_down(result, 8);
//   result += __shfl_down(result, 4);
//   result += __shfl_down(result, 2);
//   result += __shfl_down(result, 1);
//   if (threadIdx.x % 32 == 0) atomicAdd(&factor_data[blockIdx.x], result);
// }

// // Simpler version assuming dim <= 32.
// __global__ void ComputeDotProduct(double* w_data, double* x_data, double* factor_data, int dim) {
//   int result = w_data[threadIdx.x] * x_data[blockIdx.x * dim + threadIdx.x];
//   // Synchronous instructions within a warp to reduce the sum.
//   result += __shfl_down(result, 16);
//   result += __shfl_down(result, 8);
//   result += __shfl_down(result, 4);
//   result += __shfl_down(result, 2);
//   result += __shfl_down(result, 1);
//   factor_data[blockIdx.x] = result;
// }

__global__ void UpdateFactorKernel(double* factor_data, double* y_data, int samples) {
  int index = blockIdx.x * 32 + threadIdx.x;
  if (index < samples) {
    factor_data[index] = y_data[index] * (1. / (1. + exp(-y_data[index] * factor_data[index])) - 1.);
  }
}

/*
 * Use existing libraries to update the weights.
 */
void UpdateWeight(const GpuTensorStore<double, 2>& x_data,
                  const GpuTensorStore<double, 1>& y_data,
                  const GpuTensorStore<double, 1>& w_data,
                  GpuTensorStore<double, 1>* g_data) {
  const size_t dim = x_data.get_ranks()[0];
  const size_t samples = x_data.get_ranks()[1];
  g_data->Resize({dim});
  GpuTensorStore<double, 1> factor_data;
  factor_data.Resize({samples});
  double alpha = 1;
  double beta = 0;
  hipblasStatus_t return_status;

  // Three approaches: (1) gemv is 3x slower than the handwritten kernel. (2) gemm is no faster than gemv.
  // Approach 1: a hand-written kernel.
  // ComputeDotProduct<<<samples, dim>>>((double*)w_data.get_data(), (double*)x_data.get_data(), (double*)factor_data.get_data(), dim);
  // Approach 2: cuBLAS/gemv.
  return_status =
    hipblasDgemv(cublas_handle, HIPBLAS_OP_T,
                dim, samples, &alpha,
                (double*)x_data.get_data(), dim,
                (double*)w_data.get_data(), 1,
                &beta,
                (double*)factor_data.get_data(), 1);
  if (return_status != 0) {
    fprintf(stderr, "cuBLAS error %d\n", int(return_status));
  }
  // Approach 3: cuBLAS/gemm.
  // return_status =
  //  hipblasDgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
  //              1, samples, dim, &alpha,
  //              (double*)w_data.get_data(), 1,
  //              (double*)x_data.get_data(), dim,
  //              &beta,
  //              (double*)factor_data.get_data(), 1);
  // if (return_status != 0) {
  //   fprintf(stderr, "cublas error %d\n", int(return_status));
  // }

  // Manually update the factors.
  UpdateFactorKernel<<<(samples + 31) / 32, 32>>>((double*)factor_data.get_data(), (double*)y_data.get_data(), samples);
  // Use the cuBLAS library to compute the matrix multiplication.
  return_status =
    hipblasDgemv(cublas_handle, HIPBLAS_OP_N,
                dim, samples, &alpha,
                (double*)x_data.get_data(), dim,
                (double*)factor_data.get_data(), 1,
                &beta,
                (double*)g_data->get_data(), 1);
  if (return_status != 0) {
    fprintf(stderr, "cuBLAS error %d\n", int(return_status));
  }
  // Wait for GPU computations to complete.
  hipDeviceSynchronize();
}

/*
 * Compute the gradient and reduce by 32x.
 */
__global__ void ComputeGradientPart(double* x_data, double* y_data, double* w_data, int dim, int samples, int num_warps,
                                    double* interg_data) {
  int base_index = blockIdx.x * 32 + threadIdx.x;
  double factor = 0;
  // CAUTION: hard-coded dim.
  double x_data_buffer[20];
  if (base_index < samples) {
    for (int i = 0; i < dim; ++i) {
      x_data_buffer[i] = x_data[base_index * dim + i];
      factor += x_data_buffer[i] * w_data[i];
    }
    const double y_data_buffer = y_data[base_index];
    factor = y_data_buffer * (1. / (1. + exp(-y_data_buffer * factor)) - 1.);
  }
  double temp;
  for (int i = 0; i < dim; ++i) {
    temp = factor * x_data_buffer[i];
    // Every wrap of size 32 sums up the gradients sychronously. No synchronizaiton is needed.
    temp += __shfl_down(temp, 16);
    temp += __shfl_down(temp, 8);
    temp += __shfl_down(temp, 4);
    temp += __shfl_down(temp, 2);
    temp += __shfl_down(temp, 1);
    if (threadIdx.x == 0) {
      interg_data[i * num_warps + blockIdx.x] = temp;
    }
  }
}

void UpdateWeightTuned(const GpuTensorStore<double, 2>& x_data,
		const GpuTensorStore<double, 1>& y_data,
		const GpuTensorStore<double, 1>& w_data,
		GpuTensorStore<double, 1>* g_data) {
  const size_t dim = x_data.get_ranks()[0];
  const size_t samples = x_data.get_ranks()[1];
  int threads_per_block = 32;
  int num_blocks = (samples + threads_per_block - 1) / threads_per_block;
  int num_warps = (samples + 31) / 32;
  GpuTensorStore<double, 1> interg_data;
  interg_data.Resize({num_warps * dim});
  ComputeGradientPart<<<num_blocks, threads_per_block>>>(
      (double*)x_data.get_data(), (double*)y_data.get_data(), (double*)w_data.get_data(),
      dim, samples, num_warps, (double*)interg_data.get_data());
  std::vector<double> result(dim);
  for (int i = 0; i < dim; ++i) {
    // Thrust reduction is highly optimized, and this kernel is not the dominant computation part.
    result[i] = thrust::reduce(
		    thrust::device_ptr<double>((double*)interg_data.get_data() + i * num_warps),
                    thrust::device_ptr<double>((double*)interg_data.get_data() + (i+1) * num_warps),
		    0., thrust::plus<double>());
  }
  g_data->ToDevice(result);
  // Wait for GPU computations to complete.
  hipDeviceSynchronize();
}

}  // namespace app

}  // namespace canary
