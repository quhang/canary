#include "hip/hip_runtime.h"
#define EIGEN_USE_GPU

#include "cuda_helper.h"

#include <hip/hip_runtime.h>
#include "unsupported/Eigen/CXX11/Tensor"
#include "unsupported/Eigen/CXX11/Tensor"

#include "hipblas.h"

namespace canary {

hipblasHandle_t cublas_handle;
bool cublas_initialized = false;




template<typename T, size_t Dimension>
void GpuTensorStore<T, Dimension>::LoadFromHostVector(
    const std::vector<T>& input) {
  if (input.size() != get_num_elements()) {
    fprintf(stderr,
            "Deserialization for the GpuTensorStore failed internally!\n");
  } else {
    hipMemcpy(data_, input.data(), input.size() * sizeof(T),
               hipMemcpyHostToDevice);
    hipDeviceSynchronize();
  }
}

template<typename T, size_t Dimension>
void GpuTensorStore<T, Dimension>::SaveToHostVector(
    std::vector<T>* input) const {
  if (data_) {
    input->resize(get_num_elements(), 0);
    hipMemcpy(input->data(), data_, input->size() * sizeof(T),
               hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
}

template<typename T, size_t Dimension>
void GpuTensorStore<T, Dimension>::Reset() {
  if (data_) {
    hipFree(data_);
  }
  data_ = nullptr;
  true_size_ = 0;
}

template<typename T, size_t Dimension>
bool GpuTensorStore<T, Dimension>::Allocate(size_t num_elements) {
  if (true_size_ >= num_elements * sizeof(T)) {
    return true;
  } else {
    Reset();
    if (hipMalloc(&data_, num_elements * sizeof(T)) == 0) {
      true_size_ = num_elements * sizeof(T);
      return true;
    } else {
      return false;
    }
  }
}

template class GpuTensorStore<float, 1>;
template class GpuTensorStore<float, 2>;
template class GpuTensorStore<float, 3>;
template class GpuTensorStore<double, 1>;
template class GpuTensorStore<double, 2>;
template class GpuTensorStore<double, 3>;

namespace app {

Eigen::CudaStreamDevice hip_stream(0);
Eigen::GpuDevice gpu_device(&hip_stream);

void GenerateRandomData(const std::vector<double> reference,
                        GpuTensorStore<double, 2>* x_data,
                        GpuTensorStore<double, 1>* y_data) {
  if (!cublas_initialized) {
    cublas_initialized = true;
    hipblasCreate(&cublas_handle);
  }
  const size_t dim = x_data->get_ranks()[0];
  const size_t samples = x_data->get_ranks()[1];
  if (reference.size() != dim || y_data->get_ranks()[0] != samples) {
    fprintf(stderr, "Dimension mismatch for GPU execution!\n");
    return;
  }
  // Call into the Eigen library to generate random numbers.
  Eigen::TensorMap<Eigen::Tensor<double, 2>> x_tensor(
      (double*)x_data->get_data(), dim, samples);
  x_tensor.device(gpu_device) = x_tensor.random() - x_tensor.constant(0.5);

  Eigen::TensorMap<Eigen::Tensor<double, 1>> y_tensor((double*)y_data->get_data(), samples);
  GpuTensorStore<double, 1> w_data;
  w_data.ToDevice(reference);
  Eigen::TensorMap<Eigen::Tensor<double, 2>> w_tensor((double*)w_data.get_data(), dim, 1);

  // Call into the Eigen library to compute the labels.
  Eigen::array<int, 2> bcast1({1, int(samples)});
  Eigen::array<int, 1> dims1({0});
  y_tensor.device(gpu_device) =
    ((w_tensor.broadcast(bcast1) * x_tensor).sum(dims1) > y_tensor.constant(0)).select(
        y_tensor.constant(1), y_tensor.constant(-1));
  
}

void UpdateWeight(const GpuTensorStore<double, 2>& x_data,
                  const GpuTensorStore<double, 1>& y_data,
                  const GpuTensorStore<double, 1>& w_data,
                  GpuTensorStore<double, 1>* g_data) {
  const size_t dim = x_data.get_ranks()[0];
  const size_t samples = x_data.get_ranks()[1];
  Eigen::TensorMap<Eigen::Tensor<double, 2>> x_tensor((double*)(x_data.get_data()), dim, samples);
  Eigen::TensorMap<Eigen::Tensor<double, 1>> y_tensor((double*)(y_data.get_data()), samples);
  Eigen::TensorMap<Eigen::Tensor<double, 2>> w_tensor((double*)(w_data.get_data()), dim, 1);
  g_data->Resize({dim});
  Eigen::TensorMap<Eigen::Tensor<double, 1>> g_tensor((double*)(g_data->get_data()), dim);
  GpuTensorStore<double, 1> factor_data;
  factor_data.Resize({samples});
  Eigen::TensorMap<Eigen::Tensor<double, 1>> factor_tensor((double*)(factor_data.get_data()), samples);

  // Call into the Eigen library to compute the factors.
  Eigen::array<int, 2> bcast1({1, int(samples)});
  Eigen::array<int, 1> dims1({0});
  auto dot = (w_tensor.broadcast(bcast1) * x_tensor).sum(dims1);
  factor_tensor.device(gpu_device) = y_tensor * (
      y_tensor.constant(1.) / (y_tensor.constant(1.) + (-y_tensor * dot).exp())
      - y_tensor.constant(1.));
  double alpha = 1;
  double beta = 0;
  // Call into cuBLAS library to compute the matrix multiplication.
  hipblasStatus_t return_status =
    hipblasDgemv(cublas_handle, HIPBLAS_OP_N,
                dim, samples, &alpha,
                (double*)x_data.get_data(), dim,
                (double*)factor_data.get_data(), 1,
                &beta,
                (double*)g_data->get_data(), 1);
  if (return_status != 0) {
    fprintf(stderr, "cublas error %d\n", int(return_status));
  }
  hipDeviceSynchronize();
}

}  // namespace app


}  // namespace canary
